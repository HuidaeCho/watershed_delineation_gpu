#include "hip/hip_runtime.h"
#include "PathReductionGpuAlgorithm.h"


PathReductionGpuAlgorithm::PathReductionGpuAlgorithm(int blockSize):
  blockSize(blockSize)
{
}


BasinIndexMatrix PathReductionGpuAlgorithm::execute(const FlowDirectionMatrix& directionMatrix, const std::vector<CellMarker<unsigned char>>& outlet)
{
  const int height = directionMatrix.height;
  const int width = directionMatrix.width;
  const int outletCells = outlet.size();
  const unsigned int size = height * width;
  const int blocks = (size + blockSize - 1) / blockSize;
  const int outletBlocks = (outletCells + blockSize - 1) / blockSize;

  unsigned int* gpuOutletLocation = WatershedDelineationUtilities::sendOutletLocationsToGpu(outlet, width);
  unsigned char* gpuOutletLabel = WatershedDelineationUtilities::sendOutletLabelsToGpu(outlet);

  unsigned char* gpuTransferArray;
  unsigned int* gpuTargetArray;
  bool* gpuChanges;

  hipMalloc(&gpuTransferArray, size * sizeof(unsigned char));
  hipMalloc(&gpuTargetArray, size * sizeof(unsigned int));
  hipMalloc(&gpuChanges, sizeof(bool));

  FlattenedMatrix<unsigned char> transferArray = WatershedDelineationUtilities::flattenDirectionMatrixParallel(directionMatrix);
  WatershedDelineationUtilities::removeOutletDirection(transferArray, outlet);
  hipMemcpy(gpuTransferArray, transferArray.value, size * sizeof(unsigned char), hipMemcpyHostToDevice);

  directionToTargetKernel<<<blocks, blockSize>>>(gpuTransferArray, gpuTargetArray, height, width);

  bool changes;

  do
  {
    changes = false;
    hipMemcpy(gpuChanges, &changes, sizeof(bool), hipMemcpyHostToDevice);
    pathReductionKernel<<<blocks, blockSize>>>(gpuTargetArray, size, gpuChanges);
    hipMemcpy(&changes, gpuChanges, sizeof(bool), hipMemcpyDeviceToHost);
  }
  while (changes);

  clearBasinArrayKernel<<<blocks, blockSize>>>(gpuTransferArray, size);
  initializeBasinArrayKernel<<<outletBlocks, blockSize>>>(gpuTransferArray, gpuOutletLocation, gpuOutletLabel, outletCells);
  targetToBasinKernel<<<blocks, blockSize>>>(gpuTargetArray, gpuTransferArray, size);

  hipMemcpy(transferArray.value, gpuTransferArray, size * sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(gpuOutletLocation);
  hipFree(gpuOutletLabel);
  hipFree(gpuTransferArray);
  hipFree(gpuTargetArray);
  hipFree(gpuChanges);

  return WatershedDelineationUtilities::unflattenBasinMatrixParallel(transferArray);
}


__global__ void pathReductionKernel(unsigned int* targetArray, unsigned int size, bool* changes)
{
  const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  if ((index < size) && (targetArray[index] != targetArray[targetArray[index]]))
  {
    targetArray[index] = targetArray[targetArray[index]];
    *changes = true;
  }
}
