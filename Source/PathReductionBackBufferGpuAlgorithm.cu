#include "hip/hip_runtime.h"
#include "PathReductionBackBufferGpuAlgorithm.h"


PathReductionBackBufferGpuAlgorithm::PathReductionBackBufferGpuAlgorithm(int blockSize):
  blockSize(blockSize)
{
}


BasinIndexMatrix PathReductionBackBufferGpuAlgorithm::execute(const FlowDirectionMatrix& directionMatrix, const std::vector<CellMarker<unsigned char>>& outlet)
{
  const int height = directionMatrix.height;
  const int width = directionMatrix.width;
  const int outletCells = outlet.size();
  const unsigned int size = height * width;
  const int blocks = (size + blockSize - 1) / blockSize;
  const int outletBlocks = (outletCells + blockSize - 1) / blockSize;

  unsigned int* gpuOutletLocation = WatershedDelineationUtilities::sendOutletLocationsToGpu(outlet, width);
  unsigned char* gpuOutletLabel = WatershedDelineationUtilities::sendOutletLabelsToGpu(outlet);

  unsigned char* gpuTransferArray;
  unsigned int* gpuTargetReadArray;
  unsigned int* gpuTargetWriteArray;
  bool* gpuChanges;

  hipMalloc(&gpuTransferArray, size * sizeof(unsigned char));
  hipMalloc(&gpuTargetReadArray, size * sizeof(unsigned int));
  hipMalloc(&gpuTargetWriteArray, size * sizeof(unsigned int));
  hipMalloc(&gpuChanges, sizeof(bool));

  FlattenedMatrix<unsigned char> transferArray = WatershedDelineationUtilities::flattenDirectionMatrixParallel(directionMatrix);
  WatershedDelineationUtilities::removeOutletDirection(transferArray, outlet);
  hipMemcpy(gpuTransferArray, transferArray.value, size * sizeof(unsigned char), hipMemcpyHostToDevice);

  directionToTargetKernel<<<blocks, blockSize>>>(gpuTransferArray, gpuTargetReadArray, height, width);

  bool changes;

  do
  {
    changes = false;
    hipMemcpy(gpuChanges, &changes, sizeof(bool), hipMemcpyHostToDevice);
    pathReductionKernel<<<blocks, blockSize>>>(gpuTargetReadArray, gpuTargetWriteArray, size, gpuChanges);
    hipMemcpy(&changes, gpuChanges, sizeof(bool), hipMemcpyDeviceToHost);
    std::swap(gpuTargetReadArray, gpuTargetWriteArray);
  }
  while (changes);

  clearBasinArrayKernel<<<blocks, blockSize>>>(gpuTransferArray, size);
  initializeBasinArrayKernel<<<outletBlocks, blockSize>>>(gpuTransferArray, gpuOutletLocation, gpuOutletLabel, outletCells);
  targetToBasinKernel<<<blocks, blockSize>>>(gpuTargetReadArray, gpuTransferArray, size);

  hipMemcpy(transferArray.value, gpuTransferArray, size * sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(gpuOutletLocation);
  hipFree(gpuOutletLabel);
  hipFree(gpuTransferArray);
  hipFree(gpuTargetReadArray);
  hipFree(gpuTargetWriteArray);
  hipFree(gpuChanges);

  return WatershedDelineationUtilities::unflattenBasinMatrixParallel(transferArray);
}


__global__ void pathReductionKernel(unsigned int* targetReadArray, unsigned int* targetWriteArray, unsigned int size, bool* changes)
{
  const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  if ((index < size) && (targetWriteArray[index] != targetReadArray[targetReadArray[index]]))
  {
    targetWriteArray[index] = targetReadArray[targetReadArray[index]];
    *changes = true;
  }
}
