#include "hip/hip_runtime.h"
#include "LabelPullingGpuAlgorithm.h"


LabelPullingGpuAlgorithm::LabelPullingGpuAlgorithm(int blockSize):
  blockSize(blockSize)
{
}


BasinIndexMatrix LabelPullingGpuAlgorithm::execute(const FlowDirectionMatrix& directionMatrix, const std::vector<CellMarker<unsigned char>>& outlet)
{
  const int height = directionMatrix.height;
  const int width = directionMatrix.width;
  const int outletCells = outlet.size();
  const unsigned int size = height * width;
  const int blocks = (size + blockSize - 1) / blockSize;
  const int outletBlocks = (outletCells + blockSize - 1) / blockSize;

  unsigned int* gpuOutletLocation = WatershedDelineationUtilities::sendOutletLocationsToGpu(outlet, width);
  unsigned char* gpuOutletLabel = WatershedDelineationUtilities::sendOutletLabelsToGpu(outlet);

  unsigned char* gpuTransferArray;
  unsigned int* gpuTargetArray;
  bool* gpuChanges;

  hipMalloc(&gpuTransferArray, size * sizeof(unsigned char));
  hipMalloc(&gpuTargetArray, size * sizeof(unsigned int));
  hipMalloc(&gpuChanges, sizeof(bool));

  FlattenedMatrix<unsigned char> transferArray = WatershedDelineationUtilities::flattenDirectionMatrixParallel(directionMatrix);
  hipMemcpy(gpuTransferArray, transferArray.value, size * sizeof(unsigned char), hipMemcpyHostToDevice);

  directionToTargetKernel<<<blocks, blockSize>>>(gpuTransferArray, gpuTargetArray, height, width);
  clearBasinArrayKernel<<<blocks, blockSize>>>(gpuTransferArray, size);
  initializeBasinArrayKernel<<<outletBlocks, blockSize>>>(gpuTransferArray, gpuOutletLocation, gpuOutletLabel, outletCells);

  bool changes;

  do
  {
    changes = false;
    hipMemcpy(gpuChanges, &changes, sizeof(bool), hipMemcpyHostToDevice);
    labelPullingKernel<<<blocks, blockSize>>>(gpuTargetArray, gpuTransferArray, size, gpuChanges);
    hipMemcpy(&changes, gpuChanges, sizeof(bool), hipMemcpyDeviceToHost);
  }
  while (changes);

  hipMemcpy(transferArray.value, gpuTransferArray, size * sizeof(unsigned char), hipMemcpyDeviceToHost);

  hipFree(gpuOutletLocation);
  hipFree(gpuOutletLabel);
  hipFree(gpuTransferArray);
  hipFree(gpuTargetArray);
  hipFree(gpuChanges);

  return WatershedDelineationUtilities::unflattenBasinMatrixParallel(transferArray);
}


__global__ void labelPullingKernel(unsigned int* targetArray, unsigned char* basinArray, unsigned int size, bool* changes)
{
  const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < size)
  {
    if ((basinArray[index] == BASIN_NONE) && (basinArray[targetArray[index]] != BASIN_NONE))
    {
      basinArray[index] = basinArray[targetArray[index]];
      *changes = true;
    }
  }
}
